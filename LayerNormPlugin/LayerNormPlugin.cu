#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 

#include <iostream>
#include <algorithm>
#include <numeric>

#include "LayerNormPlugin.h"
#include "common.cuh"

using namespace nvinfer1;

PluginFieldCollection LayerNormPluginCreator::fc_{};
std::vector<PluginField> LayerNormPluginCreator::attr_;

/*__global__ void layerNormKernel(float *pInput, float *pOutput)*/
/*{*/
    /*const int tx = threadIdx.x, index = blockIdx.x * 256 + threadIdx.x;*/

    /*__shared__ float temp[128];*/

    /*float value0 = pInput[index];*/
    /*float value1 = pInput[index + 128];*/

    /*temp[tx] = value0 + value1;*/
    /*__syncthreads();*/

    /*for (int stride = 64; stride >= 1; stride /= 2)*/
    /*{*/
        /*if (tx < stride)*/
        /*{*/
            /*temp[tx] += temp[tx + stride];*/
        /*}*/
        /*__syncthreads();*/
    /*}*/
    /*float mean = temp[0] / 256;*/
    /*__syncthreads();*/

    /*temp[tx] = (value0 - mean) * (value0 - mean) + (value1 - mean) * (value1 - mean);*/
    /*__syncthreads();*/

    /*for (int stride = 64; stride >= 1; stride /= 2)*/
    /*{*/
        /*if (tx < stride)*/
        /*{*/
            /*temp[tx] += temp[tx + stride];*/
        /*}*/
        /*__syncthreads();*/
    /*}*/
    /*float var = temp[0] / 256;*/

    /*pOutput[index]       = (value0 - mean) * rsqrtf(var + 6e-6);*/
    /*pOutput[index + 128] = (value1 - mean) * rsqrtf(var + 6e-6);*/
/*}*/


template <typename T, unsigned TPB>
__global__ void layer_norm_kernel_small(
    const int ld, const T* input, const T* beta, const T* gamma, T* output)
{

    const T rld = T(1) / T(ld);
    const int offset = blockIdx.x * ld;

    hipcub::Sum pairSum;
    // reduce x and x^2
    kvp<T> threadData(0, 0);
    const int idx = offset + threadIdx.x;
    T val = 0;

    if (threadIdx.x < ld)
    {

        val = input[idx];

        const T rldval = rld * val;
        threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
    }

    layerNormSmall<T, T, TPB>(val, threadData, ld, idx, beta, gamma, output);
}

template <typename T, unsigned TPB>
__global__ void layer_norm_kernel(
    const int ld, const T* input, const T* beta, const T* gamma, T* output)
{
    const T rld = T(1) / T(ld);
    const int offset = blockIdx.x * ld;

    hipcub::Sum pairSum;
    // reduce x and x^2
    kvp<T> threadData(0, 0);

    for (int i = threadIdx.x; i < ld; i += TPB)
    {
        const int idx = offset + i;
        T val = T(input[idx]);

        const T rldval = rld * val;
        threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
        output[idx] = val;
    }

    layerNorm<T, T, T, TPB>(threadData, ld, offset, beta, gamma, output);
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("%f %f %f %f\n", __half2float(gamma[0]), __half2float(beta[0]), __half2float(input[0]), __half2float(output[0]));
    }
}

template <typename T>
int compute_layer_norm_tpl(hipStream_t stream, const int ld, const int n, const T* input, const T* beta,
    const T* gamma, T* output) {

    // this must be true because n is the total size of the tensor
    assert(n % ld == 0);
    const int gridSize = n / ld;
    /*constexpr int VPT = 16 / sizeof(T);*/
    if (ld <= 32) {
        constexpr int blockSize = 32;
        layer_norm_kernel_small<T, blockSize>
            <<<gridSize, blockSize, 0, stream>>>(ld, input, beta, gamma, output);
    } else if (ld <= 128) {
        constexpr int blockSize = 128;
        layer_norm_kernel_small<T, blockSize>
            <<<gridSize, blockSize, 0, stream>>>(ld, input, beta, gamma, output);
    } else if (ld <= 384) {
        constexpr int blockSize = 384;
        layer_norm_kernel_small<T, blockSize>
            <<<gridSize, blockSize, 0, stream>>>(ld, input, beta, gamma, output);
    } else {
        constexpr int blockSize = 256;
        layer_norm_kernel<T, blockSize>
            <<<gridSize, blockSize, 0, stream>>>(ld, input, beta, gamma, output);
    }
    (hipPeekAtLastError());

    return 0;
}

int compute_layer_norm(hipStream_t stream, const int ld, const int n, const float* input,
                       const float* gamma, const float* beta, float* output) {
    return compute_layer_norm_tpl<float>(stream, ld, n, input, beta, gamma, output);
}

int compute_layer_norm(hipStream_t stream, const int ld, const int n, const half* input,
                       const half* gamma, const half* beta, half* output) {
    return compute_layer_norm_tpl<half>(stream, ld, n, input, beta, gamma, output);
}

inline int64_t volume(const nvinfer1::Dims& d) {
  return std::accumulate(d.d, d.d + d.nbDims, 1, std::multiplies<int64_t>());
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    /*const int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1];*/

    /*layerNormKernel <<<nBlock, 128, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);*/

  const int input_volume = volume(inputDesc[0].dims);
  const int dim = inputDesc[0].dims.d[inputDesc[0].dims.nbDims - 1];
  const int S = input_volume / dim;

  int status = -1;

  /*const size_t word_size = getElementSize(DataType::kFLOAT);*/

  // Our plugin outputs only one tensor
  const float* input = static_cast<const float*>(inputs[0]);
  const float* gamma_ptr = static_cast<const float*>(inputs[1]);
  const float* beta_ptr = static_cast<const float*>(inputs[2]);
  float* output = static_cast<float*>(outputs[0]);

  status = compute_layer_norm(stream, dim, input_volume, input, gamma_ptr, beta_ptr, output);

    return 0;
}


REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);

